#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022 Ruilong Li, UC Berkeley.
 */

#include "include/helpers_cuda.h"

__global__ void ray_indices_kernel(
    // input
    const int n_rays,
    const int *packed_info,
    // output
    int *ray_indices)
{
    CUDA_GET_THREAD_ID(i, n_rays);

    // locate
    const int base = packed_info[i * 2 + 0];  // point idx start.
    const int steps = packed_info[i * 2 + 1]; // point idx shift.
    if (steps == 0)
        return;

    ray_indices += base;

    for (int j = 0; j < steps; ++j)
    {
        ray_indices[j] = i;
    }
}

torch::Tensor unpack_to_ray_indices(const torch::Tensor packed_info)
{
    DEVICE_GUARD(packed_info);
    CHECK_INPUT(packed_info);

    const int n_rays = packed_info.size(0);
    const int threads = 256;
    const int blocks = CUDA_N_BLOCKS_NEEDED(n_rays, threads);

    int n_samples = packed_info[n_rays - 1].sum(0).item<int>();
    torch::Tensor ray_indices = torch::zeros(
        {n_samples}, packed_info.options().dtype(torch::kInt32));

    ray_indices_kernel<<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
        n_rays,
        packed_info.data_ptr<int>(),
        ray_indices.data_ptr<int>());
    return ray_indices;
}
